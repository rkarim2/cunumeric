/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/matrix/contract.h"
#include "cunumeric/matrix/contract_template.inl"

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <hipDataType DATA_TYPE_CODE, hiptensorComputeDescriptor_t COMPUTE_TYPE_CODE, typename T>
__host__ void contract(T* lhs_data,
                       size_t lhs_ndim,
                       int64_t* lhs_shape,
                       int64_t* lhs_strides,
                       int32_t* lhs_modes,
                       const T* rhs1_data,
                       size_t rhs1_ndim,
                       int64_t* rhs1_shape,
                       int64_t* rhs1_strides,
                       int32_t* rhs1_modes,
                       const T* rhs2_data,
                       size_t rhs2_ndim,
                       int64_t* rhs2_shape,
                       int64_t* rhs2_strides,
                       int32_t* rhs2_modes)
{
  // Initialization
  auto handle      = get_cutensor();
  auto task_stream = get_cached_stream();

  // Create tensor descriptors
  hiptensorTensorDescriptor_t lhs_desc;
  hiptensorTensorDescriptor_t rhs1_desc;
  hiptensorTensorDescriptor_t rhs2_desc;
  CHECK_CUTENSOR(hiptensorInitTensorDescriptor(
    handle, &lhs_desc, lhs_ndim, lhs_shape, lhs_strides, DATA_TYPE_CODE, HIPTENSOR_OP_IDENTITY));
  CHECK_CUTENSOR(hiptensorInitTensorDescriptor(
    handle, &rhs1_desc, rhs1_ndim, rhs1_shape, rhs1_strides, DATA_TYPE_CODE, HIPTENSOR_OP_IDENTITY));
  CHECK_CUTENSOR(hiptensorInitTensorDescriptor(
    handle, &rhs2_desc, rhs2_ndim, rhs2_shape, rhs2_strides, DATA_TYPE_CODE, HIPTENSOR_OP_IDENTITY));

  // Prepare algorithm description
  uint32_t lhs_req;
  uint32_t rhs1_req;
  uint32_t rhs2_req;
  CHECK_CUTENSOR(cutensorGetAlignmentRequirement(handle, lhs_data, &lhs_desc, &lhs_req));
  CHECK_CUTENSOR(cutensorGetAlignmentRequirement(handle, rhs1_data, &rhs1_desc, &rhs1_req));
  CHECK_CUTENSOR(cutensorGetAlignmentRequirement(handle, rhs2_data, &rhs2_desc, &rhs2_req));
  cutensorContractionDescriptor_t desc;
  CHECK_CUTENSOR(cutensorInitContractionDescriptor(handle,
                                                   &desc,
                                                   &rhs1_desc,
                                                   rhs1_modes,
                                                   rhs1_req,
                                                   &rhs2_desc,
                                                   rhs2_modes,
                                                   rhs2_req,
                                                   &lhs_desc,
                                                   lhs_modes,
                                                   lhs_req,
                                                   &lhs_desc,
                                                   lhs_modes,
                                                   lhs_req,
                                                   COMPUTE_TYPE_CODE));
  cutensorContractionFind_t find;
  CHECK_CUTENSOR(cutensorInitContractionFind(handle, &find, HIPTENSOR_ALGO_DEFAULT));

  // Allocate intermediate storage
  uint64_t work_size = 0;
  CHECK_CUTENSOR(cutensorContractionGetWorkspace(
    handle, &desc, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &work_size));
  DeferredBuffer<uint8_t, 1> work_buf(Rect<1>(Point<1>(0), Point<1>(work_size - 1)),
                                      Memory::GPU_FB_MEM);
  void* work = work_buf.ptr(Point<1>(0));

  // Execute contraction
  hiptensorContractionPlan_t plan;
  CHECK_CUTENSOR(cutensorInitContractionPlan(handle, &plan, &desc, &find, work_size));
  const T alpha = 1.0;
  const T beta  = 0.0;
  CHECK_CUTENSOR(hiptensorContraction(handle,
                                     &plan,
                                     &alpha,
                                     rhs1_data,
                                     rhs2_data,
                                     &beta,
                                     lhs_data,
                                     lhs_data,
                                     work,
                                     work_size,
                                     task_stream));
}

template <>
struct ContractImplBody<VariantKind::GPU, LegateTypeCode::FLOAT_LT> {
  void operator()(float* lhs_data,
                  size_t lhs_ndim,
                  int64_t* lhs_shape,
                  int64_t* lhs_strides,
                  int32_t* lhs_modes,
                  const float* rhs1_data,
                  size_t rhs1_ndim,
                  int64_t* rhs1_shape,
                  int64_t* rhs1_strides,
                  int32_t* rhs1_modes,
                  const float* rhs2_data,
                  size_t rhs2_ndim,
                  int64_t* rhs2_shape,
                  int64_t* rhs2_strides,
                  int32_t* rhs2_modes)
  {
    contract<HIP_R_32F, HIPTENSOR_COMPUTE_DESC_32F>(lhs_data,
                                               lhs_ndim,
                                               lhs_shape,
                                               lhs_strides,
                                               lhs_modes,
                                               rhs1_data,
                                               rhs1_ndim,
                                               rhs1_shape,
                                               rhs1_strides,
                                               rhs1_modes,
                                               rhs2_data,
                                               rhs2_ndim,
                                               rhs2_shape,
                                               rhs2_strides,
                                               rhs2_modes);
  }
};

template <>
struct ContractImplBody<VariantKind::GPU, LegateTypeCode::DOUBLE_LT> {
  void operator()(double* lhs_data,
                  size_t lhs_ndim,
                  int64_t* lhs_shape,
                  int64_t* lhs_strides,
                  int32_t* lhs_modes,
                  const double* rhs1_data,
                  size_t rhs1_ndim,
                  int64_t* rhs1_shape,
                  int64_t* rhs1_strides,
                  int32_t* rhs1_modes,
                  const double* rhs2_data,
                  size_t rhs2_ndim,
                  int64_t* rhs2_shape,
                  int64_t* rhs2_strides,
                  int32_t* rhs2_modes)
  {
    contract<HIP_R_64F, HIPTENSOR_COMPUTE_DESC_64F>(lhs_data,
                                               lhs_ndim,
                                               lhs_shape,
                                               lhs_strides,
                                               lhs_modes,
                                               rhs1_data,
                                               rhs1_ndim,
                                               rhs1_shape,
                                               rhs1_strides,
                                               rhs1_modes,
                                               rhs2_data,
                                               rhs2_ndim,
                                               rhs2_shape,
                                               rhs2_strides,
                                               rhs2_modes);
  }
};

template <>
struct ContractImplBody<VariantKind::GPU, LegateTypeCode::COMPLEX64_LT> {
  void operator()(complex<float>* lhs_data,
                  size_t lhs_ndim,
                  int64_t* lhs_shape,
                  int64_t* lhs_strides,
                  int32_t* lhs_modes,
                  const complex<float>* rhs1_data,
                  size_t rhs1_ndim,
                  int64_t* rhs1_shape,
                  int64_t* rhs1_strides,
                  int32_t* rhs1_modes,
                  const complex<float>* rhs2_data,
                  size_t rhs2_ndim,
                  int64_t* rhs2_shape,
                  int64_t* rhs2_strides,
                  int32_t* rhs2_modes)
  {
    contract<HIP_C_32F, HIPTENSOR_COMPUTE_DESC_32F>(lhs_data,
                                               lhs_ndim,
                                               lhs_shape,
                                               lhs_strides,
                                               lhs_modes,
                                               rhs1_data,
                                               rhs1_ndim,
                                               rhs1_shape,
                                               rhs1_strides,
                                               rhs1_modes,
                                               rhs2_data,
                                               rhs2_ndim,
                                               rhs2_shape,
                                               rhs2_strides,
                                               rhs2_modes);
  }
};

template <>
struct ContractImplBody<VariantKind::GPU, LegateTypeCode::COMPLEX128_LT> {
  void operator()(complex<double>* lhs_data,
                  size_t lhs_ndim,
                  int64_t* lhs_shape,
                  int64_t* lhs_strides,
                  int32_t* lhs_modes,
                  const complex<double>* rhs1_data,
                  size_t rhs1_ndim,
                  int64_t* rhs1_shape,
                  int64_t* rhs1_strides,
                  int32_t* rhs1_modes,
                  const complex<double>* rhs2_data,
                  size_t rhs2_ndim,
                  int64_t* rhs2_shape,
                  int64_t* rhs2_strides,
                  int32_t* rhs2_modes)
  {
    contract<HIP_C_64F, HIPTENSOR_COMPUTE_DESC_64F>(lhs_data,
                                               lhs_ndim,
                                               lhs_shape,
                                               lhs_strides,
                                               lhs_modes,
                                               rhs1_data,
                                               rhs1_ndim,
                                               rhs1_shape,
                                               rhs1_strides,
                                               rhs1_modes,
                                               rhs2_data,
                                               rhs2_ndim,
                                               rhs2_shape,
                                               rhs2_strides,
                                               rhs2_modes);
  }
};

/*static*/ void ContractTask::gpu_variant(TaskContext& context)
{
  contract_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
